#include "hip/hip_runtime.h"
#include <torch/extension.h>

template <typename T> __global__ void cudaInverseKernel(
        const torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> sinogram,
        const torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> angles,
        const torch::PackedTensorAccessor32<T,1,torch::RestrictPtrTraits> positions,
        torch::PackedTensorAccessor32<T,4,torch::RestrictPtrTraits> image,
        const size_t batchCount,
        const size_t imageSize,
        const size_t angleCount,
        const size_t positionCount) {
    const size_t batchIdx = static_cast<size_t>(blockIdx.z*blockDim.z+threadIdx.z);
    const size_t angleIdx = static_cast<size_t>(blockIdx.y*blockDim.y+threadIdx.y);
    const size_t posIdx   = static_cast<size_t>(blockIdx.x*blockDim.x+threadIdx.x);
    if(batchIdx >= batchCount || angleIdx >= angleCount || posIdx >= positionCount) {
        return;
    }

    if(angleIdx >= imageSize || posIdx >= imageSize) {
        return;
    }
    image[batchIdx][0][angleIdx][posIdx] = posIdx+angleIdx*imageSize; //TODO
}

torch::Tensor cudaInverse(const torch::Tensor sinogram, const torch::Tensor angles, const torch::Tensor positions, const size_t imageSize) {
    const dim3 threads(8, 8, 2);
    const dim3 blocks(
        ceil(positions.sizes()[0]/static_cast<float>(threads.x)), 
        ceil(angles.sizes()[0]/static_cast<float>(threads.y)), 
        ceil(sinogram.sizes()[0]/static_cast<float>(threads.z))
    );
    torch::Tensor image = torch::zeros({1, 1, static_cast<signed long>(imageSize), static_cast<signed long>(imageSize)}, c10::TensorOptions(torch::kCUDA));
    AT_DISPATCH_FLOATING_TYPES(sinogram.scalar_type(), "radon_cudaInverse", ([&] {
            cudaInverseKernel<scalar_t><<<blocks, threads>>>(
                sinogram.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                angles.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
                positions.packed_accessor32<scalar_t,1,torch::RestrictPtrTraits>(),
                image.packed_accessor32<scalar_t,4,torch::RestrictPtrTraits>(),
                sinogram.sizes()[0],
                imageSize,
                sinogram.sizes()[3],
                sinogram.sizes()[2]
            );
        })
    );
    return image;
}